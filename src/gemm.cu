#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include<iostream>
#include<cublasXt.h>
#include<hipblas.h>
#include<hip/hip_runtime.h>

#include<cublas_utils.h>

namespace py = pybind11;

/* library to perfrom matrix multiplication */

typedef py::array_t<float> farray;
farray sgemm(farray fA,farray fB,py::array_t<int> fdevices){
	py::buffer_info bA = fA.request();
	py::buffer_info bB = fB.request();
	py::buffer_info bdevices = fdevices.request();
	
	// Assuming that A,B are 2-d arrays.

	//Create pointers in C++ for data
	float *A = static_cast<float*>(bA.ptr);
	float *B = static_cast<float*>(bB.ptr);
	int* devices = static_cast<int*>(bdevices.ptr);
	//Get shapes of A,B
	size_t M = bA.shape[0];
        size_t K = bA.shape[1];

	size_t N = bB.shape[1];

	
	

	//Have numpy allocate memory. Access that pointer.
	farray result = farray ({M,N});
	
	py::buffer_info bC = result.request();	

	float *C = static_cast<float*>(bC.ptr);
	
 	cublasXtHandle_t handle;

	CUBLAS_CHECK(cublasXtCreate(&handle));
	CUBLAS_CHECK(cublasXtDeviceSelect(handle,fdevices.size(),devices));
	float alpha = 1.0;
	float beta = 0.0;

	// performs (A@B).T
	//= B.T @ A.T
	//((A @ B ).T).T = (B.T@A.T).T
	CUBLAS_CHECK(
			cublasXtSgemm(handle,
				HIPBLAS_OP_N,HIPBLAS_OP_N,
				M,N,K,
				&alpha,
				B,N,
				A,K,
				&beta,
				C,N)
		    );



	CUBLAS_CHECK(cublasXtCreate(&handle));	
	return result;
}



typedef py::array_t<double> darray;

darray dgemm(darray fA,darray fB,py::array_t<int> fdevices){
	py::buffer_info bA = fA.request();
	py::buffer_info bB = fB.request();
	py::buffer_info bdevices = fdevices.request();
	
	// Assuming that A,B are 2-d arrays.

	//Create pointers in C++ for data
	double *A = static_cast<double*>(bA.ptr);
	double *B = static_cast<double*>(bB.ptr);
	int* devices = static_cast<int*>(bdevices.ptr);
	//Get shapes of A,B
	size_t M = bA.shape[0];
        size_t K = bA.shape[1];

	size_t N = bB.shape[1];

	
	

	//Have numpy allocate memory. Access that pointer.
	darray result = darray ({M,N});
	
	py::buffer_info bC = result.request();	

	double *C = static_cast<double*>(bC.ptr);
	
 	cublasXtHandle_t handle;

	CUBLAS_CHECK(cublasXtCreate(&handle));
	CUBLAS_CHECK(cublasXtDeviceSelect(handle,fdevices.size(),devices));
	double alpha = 1.0;
	double beta = 0.0;

	// performs (A@B).T
	//= B.T @ A.T
	//((A @ B ).T).T = (B.T@A.T).T
	CUBLAS_CHECK(
			cublasXtDgemm(handle,
				HIPBLAS_OP_N,HIPBLAS_OP_N,
				M,N,K,
				&alpha,
				B,N,
				A,K,
				&beta,
				C,N)
		    );



	CUBLAS_CHECK(cublasXtCreate(&handle));	
	return result;
}	
			
PYBIND11_MODULE(PyBlasXT,m){
	m.doc() = "testing ability to multiply in c++";
	m.def("dgemm",&dgemm,"multiplies two numpy arrays A,B  - dgemm");
	m.def("sgemm",&sgemm,"multiplies two numpy arrays A,B  - sgemm");
}
