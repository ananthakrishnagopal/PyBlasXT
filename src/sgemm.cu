#include<pybind11/pybind11.h>
#include<pybind11/numpy.h>
#include<iostream>
#include<cublasXt.h>
#include<hipblas.h>
#include<hip/hip_runtime.h>

#include<cublas_utils.h>

namespace py = pybind11;

/* library to perfrom matrix multiplication */

typedef float elementType;
typedef py::array_t<float> farray;

farray multiply(farray fA,farray fB,farray fdevices){
	py::buffer_info bA = fA.request();
	py::buffer_info bB = fB.request();
	py::buffer_info bdevices = fdevices.request();
	
	// Assuming that A,B are 2-d arrays.

	//Create pointers in C++ for data
	elementType *A = static_cast<elementType*>(bA.ptr);
	elementType *B = static_cast<elementType*>(bB.ptr);
	int* devices = static_cast<int*>(bdevices.ptr);
	//Get shapes of A,B
	size_t M = bA.shape[0];
        size_t K = bA.shape[1];

	size_t N = bB.shape[1];

	
	

	//Have numpy allocate memory. Access that pointer.
	farray result = farray ({M,N});
	
	py::buffer_info bC = result.request();	

	elementType *C = static_cast<elementType*>(bC.ptr);
	
 	cublasXtHandle_t handle;

	CUBLAS_CHECK(cublasXtCreate(&handle));
	CUBLAS_CHECK(cublasXtDeviceSelect(handle,fdevices.size(),devices));
	elementType alpha = 1.0;
	elementType beta = 0.0;

	// performs (A@B).T
	//= B.T @ A.T
	//((A @ B ).T).T = (B.T@A.T).T
	CUBLAS_CHECK(
			cublasXtSgemm(handle,
				HIPBLAS_OP_N,HIPBLAS_OP_N,
				M,N,K,
				&alpha,
				B,N,
				A,K,
				&beta,
				C,N)
		    );



	CUBLAS_CHECK(cublasXtCreate(&handle));	
	return result;
}	
			
PYBIND11_MODULE(multiply,m){
	m.doc() = "testing ability to multiply in c++";
	m.def("multiply",&multiply,"multiplies two numpy arrays A,B");
}
